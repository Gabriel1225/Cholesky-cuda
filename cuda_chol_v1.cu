#include "hip/hip_runtime.h"
# include<stdio.h>
# include<cstdio>
# include<cmath>
# include<cstdlib>
# include<ctime>
# include<time.h>
# include<hip/hip_runtime.h>
# include "chol.h"

// CUDA初始化
bool initCUDA();

// 检查是否是对角占优阵
int check_if_diagonal_dominant(const Matrix M);

// 建立由随机数组成的正定矩阵
Matrix create_positive_definite_matrix(unsigned int,unsigned int);

// host和device之间的数据传输
void copy_matrix_to_device(Matrix Mdevice,const Matrix Mhost);
void copy_matrix_from_device(Matrix Mhost,const Matrix Mdevice);

// 在GPU上初始化cholesky矩阵
Matrix allocate_matrix_on_gpu(const Matrix M);

// A被cholesky分解
Matrix A;
Matrix h_A;

// 输出矩阵元素
void print_matrix(const Matrix);

// 检查矩阵是否对称

int check_if_symmetric(const Matrix M);

// 这里选择的是输出上三角矩阵L
__global__ void chol_kernel(double * U,int ops_per_thread)
{
	// const int tid = threadIdx.x;
	// const int size = U.num_rows / THREAD_NUM;
	// Matrix L = allocate_matrix_on_gpu(U);
	// for (unsigned int i=tid*size;i<(tid+1)*size;i++)
	// {
	// 	for (unsigned int j=0;j<(i+1);j++)
	// 	{
	// 		double s = 0;
	// 		for (unsigned int k=0;k<j;k++)
	// 			s += L.elements[i*U.num_rows+k]*L.elements[j*U.num_rows+k];
	// 		L.elements[i*U.num_rows+j] = (i==j) ? sqrt(U.elements[i*U.num_rows+i]-s) : (1.0/L.elements[j*U.num_rows+j]*(U.elements[i*U.num_rows+j]-s));
	// 	}
	// }
	// return L;


	//int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = threadIdx.x;
	unsigned int i,j,k;
	unsigned int num_rows = MATRIX_SIZE;
	for (k =0;k<num_rows;k++)
	{
		if (tx == 0)
		{
			U[k*num_rows+k] = sqrt(U[k*num_rows+k]);
			for (j=(k+1);j<num_rows;j++)
			{
				U[k * num_rows + j] /=  U[k * num_rows + k];
			}
		}
		__syncthreads();

		int istart = (k+1) + tx*ops_per_thread;
		int iend = istart + ops_per_thread;
		for (i=istart;i<iend;i++)
		{	
			for (j=i;j<num_rows;j++)
				{
					U[i*num_rows+j] -= U[k*num_rows+i]*U[k*num_rows+j];
				}
		}
		__syncthreads();
	}
	__syncthreads();

	int istart = tx*ops_per_thread;
	int iend = istart+ops_per_thread;

	for(i=istart;i<iend;i++)
	{
		for (j=0;j<i;j++)
			U[i*num_rows+j] = 0.0;
	}

}


int main()
{
	if (!initCUDA())
		return 0;

	A = create_positive_definite_matrix(MATRIX_SIZE,MATRIX_SIZE);
	print_matrix(A);
	
	//int num_block = 1;
	//int threads_per_block = 512;
	int num_threads = 1;
	float ops_per_thread = MATRIX_SIZE / num_threads;
	//dim3 thread_block(threads_per_block,1,1);
	//dim3 grid(num_block,1);

	Matrix d_A = allocate_matrix_on_gpu(A);
	copy_matrix_to_device(d_A,A);
	chol_kernel<<<1,num_threads,0>>>(d_A.elements,ops_per_thread);
	hipDeviceSynchronize();
	copy_matrix_from_device(A,d_A);
	print_matrix(A);
	
	hipFree(d_A.elements);
	free(A.elements);

	return 0;
}

bool initCUDA()
{
	int count;
	hipGetDeviceCount(&count);
	if(count==0)
	{
		fprintf(stderr, "不好，没有可用的设备!\n");
		return false;
	}
	int i;
	for ( i=0;i<count;i++)
	{
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop,i)==hipSuccess)
		{
			if (prop.major >=1)
				break;
		}
	}
		if (i == count)
		{
			fprintf(stderr, "有设备，但是不支持cuda1.x以上！\n");
			return false;
		}
		hipSetDevice(i);
		return true;
}


Matrix create_positive_definite_matrix(unsigned int num_rows, unsigned int num_columns)
{
	// 配置矩阵结构体并分配元素地址空间
	Matrix M;
	M.num_columns = M.pitch = num_columns;
	M.num_rows = num_rows;
	unsigned int size = M.num_rows * M.num_columns;
	M.elements = (double*)malloc(size * sizeof(double));

	printf("正在生成 %d x %d 元素大小在正负.5之间的矩阵\n",num_rows,num_columns);
	unsigned int i;
	unsigned int j;
	for (i=0;i<size;i++)
	{
		M.elements[i] = ((double)rand()/(double)RAND_MAX)-0.5;		
	}
	printf("随机矩阵生成完成。\n");
	print_matrix(M);

	//这里使用的方法是原矩阵加上其对称阵
	printf("将矩阵转化为对称阵：\n");
	Matrix transpose;
	transpose.num_columns = transpose.pitch = num_rows;
	transpose.num_rows = num_columns;
	size = transpose.num_rows * transpose.num_columns;
	transpose.elements = (double*)malloc(size* sizeof(double));

	for (i=0;i<transpose.num_rows;i++)
		for(j=0;j<transpose.num_columns;j++)
		transpose.elements[i*transpose.num_rows + j] = M.elements[j*M.num_columns + i];

	for(i=0;i<size;i++)
		M.elements[i] += transpose.elements[i];

	printf("检查是否满足对称阵条件.......\n");
	if(check_if_symmetric(M))
		printf("满足对称阵条件\n");
	else
	{
		printf("不满足对称阵条件，程序有误！\n");
		free(M.elements);
		M.elements = NULL;
	}
	free(transpose.elements);

	printf("将对称阵转化为正定矩阵：\n");
	for (i=0;i<num_rows;i++)
		for (j=0;j<num_columns;j++)
		{
			if(i==j) 
				M.elements[i*num_rows + j] += 0.5 * M.num_rows;
		}

	if(check_if_diagonal_dominant(M))
		printf("矩阵是正定阵，满足条件\n");
	else
	{
		printf("矩阵不是正定阵，程序有误\n");
		free(M.elements);
		M.elements = NULL;
	}
	return M;
}


int check_if_diagonal_dominant(const Matrix M)
{
	double diag_element;
	double sum;
	for (unsigned int i=0;i<M.num_rows;i++)
	{	
		diag_element = M.elements[i*M.num_rows+i];
		sum = 0.0;
		for(unsigned int j=0;j<M.num_columns;j++)
		{ 
			if(i != j) sum += abs(M.elements[i*M.num_rows+j]);
		}	
		if (diag_element < sum) return 0; 
	}
	return 1;
}

void copy_matrix_to_device(Matrix Mdevice,const Matrix Mhost)
{
	Mdevice.num_rows = Mhost.num_rows;
	Mdevice.num_columns = Mhost.num_columns;
	Mdevice.pitch = Mhost.pitch;
	int size = Mhost.num_rows * Mhost.num_columns;
	//hipMalloc((void**)&Mdevice.elements,sizeof(double)* size);
	hipMemcpy(Mdevice.elements,Mhost.elements,sizeof(double)* size,hipMemcpyHostToDevice);
	printf("矩阵从host到device传输完成！\n");
}


void copy_matrix_from_device(Matrix Mhost,Matrix Mdevice)
{
	Mhost.num_rows = Mdevice.num_rows;
	Mhost.num_columns = Mdevice.num_columns;
	int size = Mdevice.num_rows*Mdevice.num_columns;
	hipMemcpy(Mhost.elements,Mdevice.elements,sizeof(double)*size,hipMemcpyDeviceToHost);
	printf("矩阵从device到host传输完成!\n");
}

Matrix allocate_matrix_on_gpu(const Matrix M)
{
	Matrix L;
	L.num_rows = M.num_rows;
	L.num_columns = L.pitch = M.num_columns;
	int size = L.num_rows * L.num_columns;
	hipMalloc((void**)&L.elements,sizeof(double)*size);
	return L;
}

void print_matrix(const Matrix M)
{
	for (unsigned int i = 0; i<M.num_rows; i++)
	{
		for(unsigned int j = 0;j<M.num_columns;j++)	
			printf("%f ",M.elements[i*M.num_rows+j]);
	printf("\n");
	}
	printf("\n");
}

int check_if_symmetric(const Matrix M)
{
	for (unsigned int i=0;i<M.num_rows;i++)
		for(unsigned int j=0;j<M.num_columns;j++)
		{
			if (M.elements[i*M.num_rows+j] != M.elements[j*M.num_columns+i])
				return 0;
		}
		return 1;
}









